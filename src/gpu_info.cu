
#include <hip/hip_runtime.h>
#include <iostream>

int _ConvertSMVer2Cores(int major, int minor) {
    struct SMtoCores {
        int SM;
        int Cores;
    };

    // Table of SM versions
    static const SMtoCores SM_Cores_Table[] = {
        {0x30, 192},  // Kepler (3.x): 192 cores/SM
        {0x32, 192},  // Kepler (3.2): 192 cores/SM
        {0x35, 192},  // Kepler (3.5): 192 cores/SM
        {0x50, 128},  // Maxwell (5.0): 128 cores/SM
        {0x52, 128},  // Maxwell (5.2): 128 cores/SM
        {0x60, 64},   // Pascal (6.0): 64 cores/SM
        {0x61, 128},  // Pascal (6.1): 128 cores/SM
        {0x70, 64},   // Volta (7.0): 64 cores/SM
        {0x75, 64},   // Turing (7.5): 64 cores/SM
        {0x80, 64},   // Ampere (8.0): 64 cores/SM
        {0x86, 128},  // Ampere (8.6): 128 cores/SM
        {-1, -1}      // End marker
    };

    int sm_version = (major << 4) + minor;
    for (int i = 0; SM_Cores_Table[i].SM != -1; i++) {
        if (SM_Cores_Table[i].SM == sm_version) {
            return SM_Cores_Table[i].Cores;
        }
    }

    std::cerr << "Unknown SM version: " << major << "." << minor << "\n";
    return 0;
}

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for (int device = 0; device < deviceCount; device++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, device);

        std::cout << "Device " << device << ": " << prop.name << "\n";
        std::cout << "  Compute capability: " << prop.major << "." << prop.minor << "\n";
        std::cout << "  Total Global Memory: " << prop.totalGlobalMem / (1024 * 1024) << " MB\n";
        std::cout << "  Multiprocessors: " << prop.multiProcessorCount << "\n";
        std::cout << "  CUDA Cores/MP: " << _ConvertSMVer2Cores(prop.major, prop.minor) << "\n";
        std::cout << "  Total CUDA Cores: " << prop.multiProcessorCount * _ConvertSMVer2Cores(prop.major, prop.minor) << "\n";
        std::cout << "  Max Threads/Block: " << prop.maxThreadsPerBlock << "\n";
        std::cout << "  Max Grid Size: (" << prop.maxGridSize[0] << ", " << prop.maxGridSize[1] << ", " << prop.maxGridSize[2] << ")\n";
        std::cout << "  Max Threads/Dimension: (" << prop.maxThreadsDim[0] << ", " << prop.maxThreadsDim[1] << ", " << prop.maxThreadsDim[2] << ")\n";
    }

    return 0;
}