
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <hipblas.h>

#ifndef SIZE
#define SIZE 4096
#endif
#ifndef MAX_SIZE
#define MAX_SIZE 4096
#endif
#ifndef NRUNS
#define NRUNS 30
#endif
#ifndef MATRIX_TYPE
#define MATRIX_TYPE int
#endif

typedef MATRIX_TYPE *Matrix;
const size_t N = SIZE;
double MULT_TIMES[NRUNS];
struct timeval start, end;

// CUDA API error checking macro
#define cudaCheck(error)                     \
    if (error != hipSuccess)                \
    {                                        \
        printf("Fatal error: %s at %s:%d\n", \
               hipGetErrorString(error),    \
               __FILE__, __LINE__);          \
        exit(1);                             \
    }

double multiply_matrices(const Matrix A, const Matrix B, Matrix C, hipblasHandle_t handle)
{
    size_t size = N * N * sizeof(int);
    Matrix Ad, Bd, Cd;
    hipMalloc(&Ad, size);
    hipMalloc(&Bd, size);
    hipMalloc(&Cd, size);

    hipMemcpy(Ad, A, size, hipMemcpyHostToDevice);
    hipMemcpy(Bd, B, size, hipMemcpyHostToDevice);

    dim3 dBlock(32, 32);
    dim3 dGrid(N / 32, N / 32);

    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipEventRecord(startEvent, 0);

    const MATRIX_TYPE alpha = 1.0;
    const MATRIX_TYPE beta = 0.0;

    if (sizeof(MATRIX_TYPE) == sizeof(double))
    {
        // Double precision
        hipblasStatus_t stat = hipblasDgemm(
            handle,
            HIPBLAS_OP_T, HIPBLAS_OP_T,
            N, N, N,
            (const double *)&alpha,
            (const double *)Ad, N,
            (const double *)Bd, N,
            (const double *)&beta,
            (double *)Cd, N);
        if (stat != HIPBLAS_STATUS_SUCCESS)
        {
            fprintf(stderr, "hipblasDgemm failed\n");
            exit(1);
        }
    }
    else if (sizeof(MATRIX_TYPE) == sizeof(float))
    {
        // Single precision
        hipblasStatus_t stat = hipblasSgemm(
            handle,
            HIPBLAS_OP_T, HIPBLAS_OP_T,
            N, N, N,
            (const float *)&alpha,
            (const float *)Ad, N,
            (const float *)Bd, N,
            (const float *)&beta,
            (float *)Cd, N);
        if (stat != HIPBLAS_STATUS_SUCCESS)
        {
            fprintf(stderr, "hipblasSgemm failed\n");
            exit(1);
        }
    }
    else
    {
        fprintf(stderr, "Unsupported MATRIX_TYPE. Use float or double.\n");
        exit(1);
    }

    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);

    float milliseconds = 0.0f;
    hipEventElapsedTime(&milliseconds, startEvent, stopEvent);

    hipMemcpy(C, Cd, size, hipMemcpyDeviceToHost);
    hipFree(Ad);
    hipFree(Bd);
    hipFree(Cd);

    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    return milliseconds / 1000;
}

double total_time(struct timeval start, struct timeval end)
{
    return (end.tv_sec - start.tv_sec) + (end.tv_usec - start.tv_usec) / 1e6;
}

double read_matrix(FILE *file, Matrix arr, size_t n, size_t start_element)
{
    gettimeofday(&start, NULL);

    size_t offset = start_element * sizeof(int);
    fseek(file, offset, SEEK_SET);
    fread(arr, sizeof(int), n * n, file);

    gettimeofday(&end, NULL);
    return total_time(start, end);
}

double print_matrix(FILE *file, Matrix C, size_t n)
{
    gettimeofday(&start, NULL);

    const char *format = (sizeof(n) == sizeof(int)) ? "%d " : "%.2f ";
    for (size_t i = 0; i < n; i++)
    {
        for (size_t j = 0; j < n; j++)
        {
            fprintf(file, format, C[i * n + j]);
        }
        fprintf(file, "\n");
    }

    gettimeofday(&end, NULL);
    return total_time(start, end);
}

int main(int argc, char *argv[])
{
    const char *matrix_file_name = argv[1];
    const char *time_log_name = argv[2];
    const char *result_log_name = argv[3];

    double read_time = 0.0, write_time = 0.0, avg_mult_time = 0.0;

    Matrix A, B, C;
    A = (MATRIX_TYPE *)malloc(N * N * sizeof(MATRIX_TYPE));
    B = (MATRIX_TYPE *)malloc(N * N * sizeof(MATRIX_TYPE));
    C = (MATRIX_TYPE *)malloc(N * N * sizeof(MATRIX_TYPE));

    FILE *matrix_file = fopen(matrix_file_name, "rb");
    read_time = read_matrix(matrix_file, A, N, 0) + read_matrix(matrix_file, B, N, N * N);
    fclose(matrix_file);

    // Initialize cublas
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    for (int i = 0; i <= NRUNS; i++)
    {
        if (i > 0)
            MULT_TIMES[i - 1] = multiply_matrices(A, B, C, handle);
        else
            multiply_matrices(A, B, C, handle); // Warm up
    }

    hipblasDestroy(handle);

    FILE *result_log = fopen(result_log_name, "a");
    write_time = print_matrix(result_log, C, N);
    fclose(result_log);

    for (int i = 0; i < NRUNS; i++)
    {
        avg_mult_time += MULT_TIMES[i];
    }
    avg_mult_time /= NRUNS;

    FILE *time_log = fopen(time_log_name, "a");
    fprintf(time_log, "Read time: %.8f seconds\nWrite time: %.8f seconds\nMultiplication time (avg): %.8f seconds\n", read_time, write_time, avg_mult_time);
    fclose(time_log);

    free(A);
    free(B);
    free(C);

    return 0;
}