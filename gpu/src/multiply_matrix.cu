#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>

#ifndef SIZE
#define SIZE 4096
#endif
#ifndef MAX_SIZE
#define MAX_SIZE 4096
#endif
#ifndef NRUNS
#define NRUNS 30
#endif
#ifndef MATRIX_TYPE
#define MATRIX_TYPE int
#endif
// todo how to define best n?
// todo how are n blocks being assigned, isnt it through job?
#ifndef TILE_WIDTH
#define TILE_WIDTH 32
#endif

typedef MATRIX_TYPE *Matrix;
const int N = SIZE;
double MULT_TIMES[NRUNS];
struct timeval start, end;

// CUDA API error checking macro
#define cudaCheck(error)                     \
    if (error != hipSuccess)                \
    {                                        \
        printf("Fatal error: %s at %s:%d\n", \
               hipGetErrorString(error),    \
               __FILE__, __LINE__);          \
        exit(1);                             \
    }

// todo why isnt matrix A and B const anymore?
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
    __shared__ MATRIX_TYPE Ads[TILE_WIDTH][TILE_WIDTH];
    __shared__ MATRIX_TYPE Bds[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;
    MATRIX_TYPE Cval = 0;

    // loop over tiles
    for (int m = 0; m < N / TILE_WIDTH; m++)
    {
        Ads[ty][tx] = A[row * N + m * TILE_WIDTH + tx];
        Bds[ty][tx] = B[(m * TILE_WIDTH + ty) * N + col];
        __syncthreads();

        // loop over elements in tile
        for (int k = 0; k < TILE_WIDTH; k++)
        {
            Cval += Ads[ty][k] * Bds[k][tx];
        }
        __syncthreads();
    }
    C[row * N + col] = Cval; // write to global memory
}

double multiply_matrices(const Matrix A, const Matrix B, Matrix C)
{
    int size = N * N * sizeof(int);
    Matrix Ad, Bd, Cd;
    hipMalloc(&Ad, size);
    hipMalloc(&Bd, size);
    hipMalloc(&Cd, size);

    hipMemcpy(Ad, A, size, hipMemcpyHostToDevice);
    hipMemcpy(Bd, B, size, hipMemcpyHostToDevice);

    dim3 dBlock(32, 32);
    dim3 dGrid(N / 32, N / 32);

    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    hipEventRecord(startEvent, 0);
    MatMulKernel<<<dGrid, dBlock>>>(Ad, Bd, Cd);
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);

    double milliseconds = 0.0;
    hipEventElapsedTime(&milliseconds, startEvent, stopEvent);

    hipMemcpy(C, Cd, size, hipMemcpyDeviceToHost);
    hipFree(Ad);
    hipFree(Bd);
    hipFree(Cd);

    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    return milliseconds / 1000;
}

double total_time(struct timeval start, struct timeval end)
{
    return (end.tv_sec - start.tv_sec) + (end.tv_usec - start.tv_usec) / 1e6;
}

double read_matrix(FILE *file, Matrix arr, int n, size_t start_element)
{
    gettimeofday(&start, NULL);

    size_t offset = start_element * sizeof(int);
    fseek(file, offset, SEEK_SET);
    fread(arr, sizeof(int), n * n, file);

    gettimeofday(&end, NULL);
    return total_time(start, end);
}

double print_matrix(FILE *file, Matrix C, int n)
{
    gettimeofday(&start, NULL);

    const char *format = (sizeof(n) == sizeof(int)) ? "%d " : "%.2f ";
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            fprintf(file, format, C[i * n + j]);
        }
        fprintf(file, "\n");
    }

    gettimeofday(&end, NULL);
    return total_time(start, end);
}

int main()
{
    const char *matrix_file_name = argv[1];
    const char *time_log_name = argv[2];
    const char *result_log_name = argv[3];

    double read_time = 0.0, write_time = 0.0, avg_mult_time = 0.0;

    Matrix A, B, C;
    A = (MATRIX_TYPE *)malloc(N * N * sizeof(MATRIX_TYPE));
    B = (MATRIX_TYPE *)malloc(N * N * sizeof(MATRIX_TYPE));
    C = (MATRIX_TYPE *)malloc(N * N * sizeof(MATRIX_TYPE));

    FILE *matrix_file = fopen(matrix_file_name, "rb");
    read_time = read_matrix(matrix_file, A, N, 0) + read_matrix(matrix_file, B, N, N * N);
    fclose(matrix_file);

    for (int i = 0; i <= NRUNS; i++)
    {
        if (i > 0)
            MULT_TIMES[i - 1] = multiply_matrices(A, B, C);
        else
            multiply_matrices(A, B, C); // Warm up
    }

    FILE *result_log = fopen(result_log_name, "a");
    write_time = print_matrix(result_log, C, N);
    fclose(result_log);

    for (int i = 0; i < NRUNS; i++)
    {
        avg_mult_time += MULT_TIMES[i];
    }
    avg_mult_time /= NRUNS;

    FILE *time_log = fopen(time_log_name, "a");
    fprintf(time_log, "Read time: %.8f seconds\nWrite time: %.8f seconds\nMultiplication time (avg): %.8f seconds\n", read_time, write_time, avg_mult_time);
    fclose(time_log);

    free(A);
    free(B);
    free(C);

    return 0;
}